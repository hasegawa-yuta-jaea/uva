#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <iomanip>
#include <limits>

#define GPU
#include "utility/CUDA_SAFE_CALL.cuda.h"

#include "util/timer.hpp"
#include "util/signal.hpp"

const int num_gpu = 16;
const long elem = (num_gpu *1024l*1024l*1024l /8) *16; // weak scaling in GiB
const int iter = 4;

const int nth = 1024;
const long grid = elem/nth;
const long block = nth;

__device__ __forceinline__ long index(const int gpu) { 
  return threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x*gpu*0); // gridDim.x*gpu in nvidia document, but slow when total mem > 32 GB??
}

__device__ __forceinline__ long index_boundary(const long idx, const int gpu) {
  if(gpu == 0 && idx < 0) return idx + elem;
  else if(gpu == num_gpu-1 && idx >= elem/num_gpu) return idx - elem;
  else return idx;
}

__global__ void init(float* dst, float* src, const long ofs, const int gpu) {
  const long k = index(gpu);
  dst[k] = src[k] = k;
}

__global__ void foo(float *const dst, const float *const src, const long ofs, const int gpu) {
  const long idx = index(gpu);
  //// stream test
  //dst[idx] = src[idx];
  //// 1D diffusion; 3 stencil; periodic boundary
  //const float c = 0.01f;
  //const long im = index_boundary(idx-1, gpu);
  //const long ip = index_boundary(idx+1, gpu);
  //dst[idx] = (1.f - 2.f*c)*src[idx] + c*(src[im] + src[ip]);
  // 1D shift test
  const long far = 1;
  const long isx = index_boundary(idx-far);
  dst[idx] = src[isx];
}

int main(int argc, char** argv) try {
  util::timer timer;
  util::signal signal(SIGINT);
  float *dst, *src;

  std::cout << "step: malloc-gpu" << std::endl;
  timer.elapse("malloc-gpu", [&]() {
    hipMallocManaged(&dst, elem * sizeof(float));
    hipMallocManaged(&src, elem * sizeof(float));
  });

  if(std::numeric_limits<int>::max() < elem) { 
    std::cerr << "warning: huge number of elems, which needs long (int will bug)" << std::endl; 
    std::cerr << " int max = " << std::numeric_limits<int>::max() << ", #elems = " << elem << std::endl;
  }

  std::cout << "total mem = " << 2l*elem*sizeof(float) / 1024/1024/1024 << " GiB" << std::endl;
  std::cout << "  per gpu = " << 2l*elem*sizeof(float) / 1024/1024/1024./num_gpu << " GiB" << std::endl;

  std::cout << "step: init" << std::endl;
  timer.elapse("init-gpu", [&]() {
    for(int i=0; i<num_gpu; i++) {
      hipSetDevice(i);
      const long ofs = elem*i/num_gpu; // ofs = 0 in nvidia's document, but slow when total mem > 32 GB??
      init<<<grid/num_gpu, block>>>(dst + ofs, src + ofs, ofs, i);
    }
    for(int i=0; i<num_gpu; i++) {
      hipSetDevice(i);
      CUDA_SAFE_CALL(hipDeviceSynchronize());
    }
  });

  for(int i=0; i<num_gpu; i++) {
    size_t mfree, mtotal;
    CUDA_SAFE_CALL(hipSetDevice(i));
    CUDA_SAFE_CALL(hipMemGetInfo(&mfree, &mtotal));
    std::cout << "gpu " << std::setw(2) << std::setfill(' ') << i << ": "
      << std::setw(4) << double(mtotal - mfree) /1024./1024./1024. << " GiB used" << std::endl;
  }

  std::cout << "step: foo (iterative)" << std::endl;
  timer.elapse("foo", [&]() {
    double bw_max = 0;
    while(!signal) {
      util::timer timer;
      timer.elapse("foo", [&]() {
        for(int t=0; t<iter; t++) {
          float* tmp = src;
          src = dst;
          dst = tmp;
          for(int i=0; i<num_gpu; i++) {
            hipSetDevice(i);
            const long ofs = elem*i/num_gpu; // ofs = 0 in nvidia's document, but slow when total mem > 32 GB??
            foo<<<grid/num_gpu, block>>>(dst + ofs, src + ofs, ofs, i);
          }
          for(int i=0; i<num_gpu; i++) {
            hipSetDevice(i);
            CUDA_SAFE_CALL(hipDeviceSynchronize());
          }
        }
      });
      const double bw_cache = 2.* elem* sizeof(float)* iter / timer["foo"] / 1024. / 1024. / 1024.;
      bw_max = std::max(bw_max, bw_cache);
      std::cout << "bandwidth: " << bw_max << " GiB/s max, " << bw_cache << " GiB/s recent\r" << std::flush;
    }
  });
  std::cout << std::endl << "keyboard interrupted, finish calculation" << std::endl;

  std::cout << std::endl;
  for(int i=0; i<num_gpu; i++) {
    size_t mfree, mtotal;
    CUDA_SAFE_CALL(hipSetDevice(i));
    CUDA_SAFE_CALL(hipMemGetInfo(&mfree, &mtotal));
    std::cout << "gpu " << std::setw(2) << std::setfill(' ') << i << ": "
      << std::setw(4) << double(mtotal - mfree) /1024./1024./1024. << " GiB used" << std::endl;
  }

  timer.elapse("fina-GPU", hipDeviceReset);

  timer.showall();

  return 0;
} catch (const Utility::Exception& e) {
  std::cerr << "fatal: " << e.ToString() << std::endl;
  return 1;
} catch (...) {
  std::cerr << "fatal: unknown error" << std::endl;
  return 2;
}
