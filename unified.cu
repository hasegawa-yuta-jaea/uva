#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include <iostream>
#include <iomanip>
#include <limits>

#define GPU
#include "util/timer.hpp"
#include "util/cu_vector.hpp"
#include "util/cuda_safe_call.hpp"

// #include <numeric> // c++17 gcd
#include <type_traits>
namespace std {
  template<typename T, typename U> 
  constexpr typename common_type<T, U>::type 
  gcd(const T& a, const U& b) {
    return b==0 ? a : gcd(b, a%b);
  }
  template<typename T, typename U>
  constexpr typename common_type<T, U>::type
  max(const T& a, const U& b) {
    return a<b ? b : a;
  }
  template<typename T, typename U>
  constexpr typename common_type<T, U>::type
  min(const T& a, const U& b) {
    return a<b ? a : b;
  }
}

using real = float;

// multi-gpu
constexpr int gx { 16 };
constexpr int gy { 1  };
constexpr int gz { 1  };
constexpr int num_gpu { gx*gy*gz };
constexpr int gpu[] { 0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15 };

// grid
constexpr bool strong { false };
constexpr long nx_ { 1024 / gx * (strong ? 1 : gx) }; // if strong scaling, devide by g{x,y,z}
constexpr long ny_ { 1024 / gy * (strong ? 1 : gy) };
constexpr long nz_ { 1024 / gz * (strong ? 1 : gz) };
template<bool Cond, class Then, class Else> struct if_ { using type = Then; };
template<class Then, class Else> struct if_<false, Then, Else> { using type = Else; };
template<long L> struct enough_type { using type = 
  typename if_<(L > std::numeric_limits<int>::max()), long, int>::type;
};
using aint = typename enough_type<nx_ * ny_ * nz_ * gx * gy * gz>::type;
//using aint = long;

constexpr aint nx { nx_ };
constexpr aint ny { ny_ }; 
constexpr aint nz { nz_ };
constexpr aint NX { nx*gx };
constexpr aint NY { ny*gy };
constexpr aint NZ { nz*gz };
constexpr aint elem { NX*NY*NZ };

// gpu kernel
constexpr int nth { 256 };
constexpr int tx { std::gcd(256, nx) };
constexpr int ty { nth/tx };
constexpr int tz { nth/tx/ty };
static_assert(nth == tx*ty*tz, "check blockDim.{x,y,z}");

// measure
constexpr int iter { 2 };
constexpr int iiter { 10 };

template<class Func, class... Args> __global__ void kernel(Func func, Args... args) { func(args...); }

int main(int argc, char** argv) try {
  static_cast<void>(argc);
  static_cast<void>(argv);
  util::timer timer;
  util::cu_ptr<real> src(elem), dst(elem);
  //for(aint i=0; i<elem; i++) {
  //  src[i] = dst[i] = i;
  //}
  //std::cout << src.size() << std::endl;
  //util::cu_vector<real> src, dst;
  //for(aint i=0; i<elem; i++) {
  //  src.push_back(real(i));
  //  dst.push_back(real(2*i));
  //}

  //// test task id
  //constexpr int lv_max = 3;
  //constexpr int num_color = 8;
  //util::cu_vec3d<int, lv_max, num_color> task_id;
  //for(int lv=0; lv<lv_max; lv++) {
  //  for(int color=0; color<num_color; color++) {
  //    for(int i=0; i<(1+lv*color); i++) {
  //      task_id[lv][color].push_back(2*color + 100.f*lv);
  //    }
  //  }
  //}
  //for(int lv=0; lv<lv_max; lv++) {
  //  for(int color=0; color<num_color; color++) {
  //    std::cout << "task_id[" << lv << "][" << color << "] = " << std::flush;
  //    for(const auto& id: task_id[lv][color]) {
  //      std::cout << id << " " << std::flush;
  //    }
  //    std::cout << std::endl;
  //  }
  //}

  std::cout << " aint = " << typeid(aint).name() << std::endl;

  std::cout << "total mem = " << 2l*elem*sizeof(real) / 1024/1024/1024. << " GiB" << std::endl;
  std::cout << "  per gpu = " << 2l*elem*sizeof(real) / 1024/1024/1024./num_gpu << " GiB" << std::endl;
  std::cout << "total mesh = (" << NX << ", " << NY << ", " << NZ << ")" << std::endl;
  std::cout << "  partition= (" << gx << ", " << gy << ", " << gz << ")" << std::endl;
  std::cout << "  per gpu  = (" << nx << ", " << ny << ", " << nz << ")" << std::endl;

  std::cout << "step: init" << std::endl;
  timer.elapse("init", [&]() {
    const size_t memall = elem * sizeof(real);
    const size_t memgpu = memall / num_gpu;
    //hipMallocManaged(&dst.data(), memall);
    //hipMallocManaged(&src.data(), memall);
    for(int i=0; i<num_gpu; i++) {
      hipMemAdvise(dst.data(), memall, hipMemAdviseSetAccessedBy, gpu[i]);
      hipMemAdvise(src.data(), memall, hipMemAdviseSetAccessedBy, gpu[i]);
    }
    for(int i=0; i<num_gpu; i++) {
      const size_t ofs = elem*i/num_gpu;
      hipMemAdvise(dst.data() + ofs, memgpu, hipMemAdviseSetPreferredLocation, gpu[i]);
      hipMemAdvise(src.data() + ofs, memgpu, hipMemAdviseSetPreferredLocation, gpu[i]);
      hipMemPrefetchAsync(dst.data() + ofs, memgpu, gpu[i]);
      hipMemPrefetchAsync(src.data() + ofs, memgpu, gpu[i]);
    }
    for(int gi=0; gi<num_gpu; gi++) {
      hipSetDevice(gpu[gi]);
      kernel<<<elem/num_gpu/nth, nth>>>(
        [=]__device__(real* buf1, real* buf2) {
          const aint ijk = threadIdx.x + blockIdx.x*blockDim.x + gi*blockDim.x*gridDim.x;
          buf1[ijk] = buf2[ijk] = ijk;
        }, dst.data(), src.data()
      );
    }
    for(int i=0; i<num_gpu; i++) {
      hipSetDevice(gpu[i]);
      CUDA_SAFE_CALL(hipDeviceSynchronize());
    }
  });

  for(int i=0; i<num_gpu; i++) {
    size_t mfree, mtotal;
    CUDA_SAFE_CALL(hipSetDevice(i));
    CUDA_SAFE_CALL(hipMemGetInfo(&mfree, &mtotal));
    std::cout << "gpu " << std::setw(2) << std::setfill(' ') << i << ": "
      << std::setw(4) << double(mtotal - mfree) /1024./1024./1024. << " GiB used" << std::endl;
  }

  hipProfilerStart();
  std::cout << "step: foo (iterative)" << std::endl;
  timer.elapse("foo", [&]() {
    double bw_max = 0;
    for(int tt=0; tt<iiter; tt++) {
      util::timer timer;
      if(tt % (iiter/2) == 0) { // retouch
        for(aint i=0; i<elem; i++) {
          src[i] = dst[i] = i;
        }
      }
      timer.elapse("foo", [&]() {
        for(int t=0; t<iter; t++) {
          dst.swap(src);
          //for(int i=0; i<num_gpu; i++) {
          for(int gk=0; gk<gz; gk++) for(int gj=0; gj<gy; gj++) for(int gi=0; gi<gx; gi++) {
            hipSetDevice(gpu[gi + gx*(gj + gy*gk)]);
            kernel<<<dim3(nx/tx, ny/ty, nz/tz), dim3(tx, ty, tz)>>>(
              [=]__device__(real* buf1, const real* buf2) {
                const aint i = threadIdx.x + blockIdx.x*blockDim.x;
                const aint j = threadIdx.y + blockIdx.y*blockDim.y;
                const aint k = threadIdx.z + blockIdx.z*blockDim.z;
                auto& I = gi;
                auto& J = gj;
                auto& K = gk;
                aint im = i-1, ip = i+1, jm = j-1, jp = j+1, km = k-1, kp = k+1;
                aint IM = I, IP = I, JM = J, JP = J, KM = K, KP = K;
                if(im < 0) { im = nx-1; IM = (I-1+gx)%gx; }
                if(ip >= nx) { ip = 0; IP = (I+1)%gx; }
                if(jm < 0) { jm = ny-1; JM = (J-1+gy)%gy; } 
                if(jp >= ny) { jp = 0; JP = (J+1)%gy; }
                if(km < 0) { km = nz-1; KM = (K-1+gz)%gz; }
                if(kp >= nz) { kp = 0; KP = (K+1)%gz; }
                auto idx = []__device__(aint i, aint j, aint k, aint I, aint J, aint K) {
                  return i + nx*(j + ny*(k + nz*(I + gx*(J + gy*K))));
                };
                const aint ijk = idx(i, j, k, I, J, K);
                const aint je[6] = { idx(im, j, k, IM, J, K),
                                 idx(ip, j, k, IP, J, K),
                                 idx(i, jm, k, I, JM, K),
                                 idx(i, jp, k, I, JP, K),
                                 idx(i, j, km, I, J, KM),
                                 idx(i, j, kp, I, J, KP) };
                const real cc = 0.1f;
                buf1[ijk] = (1.f-6.f*cc)*buf2[ijk] + cc*(buf2[je[0]]);;// + buf2[je[1]] + buf2[je[2]] + buf2[je[3]] + buf2[je[4]] +buf2[je[5]]);
              }, dst.data(), src.data()
            );
          }
          for(int i=0; i<num_gpu; i++) {
            hipSetDevice(gpu[i]);
            CUDA_SAFE_CALL(hipDeviceSynchronize());
          }
        }
      });
      const double bw_cache = 2.* elem* sizeof(real)* iter / timer["foo"] / 1024. / 1024. / 1024.;
      bw_max = std::max(bw_max, bw_cache);
      std::cout << "bandwidth: " << bw_max << " GiB/s max, " << bw_cache << " GiB/s recent\r" << std::endl;
    }
  });

  std::cout << std::endl;
  for(int i=0; i<num_gpu; i++) {
    size_t mfree, mtotal;
    CUDA_SAFE_CALL(hipSetDevice(i));
    CUDA_SAFE_CALL(hipMemGetInfo(&mfree, &mtotal));
    std::cout << "gpu " << std::setw(2) << std::setfill(' ') << i << ": "
      << std::setw(4) << double(mtotal - mfree) /1024./1024./1024. << " GiB used" << std::endl;
  }

  for(aint max=0, min=std::numeric_limits<aint>::max(), i=0; i<elem; i++) {
    max = std::max(max, dst[i]);
    min = std::min(min, dst[i]);
    if(i == elem-1) { std::cout << "dst = " << min << " -- " << max << std::endl; }
  }

  timer.elapse("fina-GPU", []() {
    hipProfilerStop();
    hipDeviceReset();
  });

  timer.showall();

  return 0;
} catch (const std::runtime_error& e) {
  std::cerr << "fatal: " << e.what() << std::endl;
  return 1;
} catch (...) {
  std::cerr << "fatal: unknown error" << std::endl;
  return 2;
}
